#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>
#include <assert.h>

//#define N_BLOCK 16
#define nn 512

#include "inc_precision.h"

struct timeval start, stop;
int64_t sec    ;
int64_t usec   ;
double elapsed ;

REAL a[nn][nn];       /** matrixes**/
REAL b[nn][nn];
REAL c[nn][nn];
REAL check[nn][nn];

/*---------------------------------------------------------*/


__global__ void gpu_mm(REAL* d_a, REAL* d_b, REAL* d_c, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    REAL tmp = 0;

    for (int k = 0; k < n; ++k) {
        REAL d_a_tile = d_a[row*n+k];
        REAL d_b_tile = d_b[k*n+col];
        tmp += d_a_tile * d_b_tile;
    }
    d_c[row*nn+col] = tmp;
}

/*---------------------------------------------------------*/

int main()
{
  int i, j, k;
  int N_BLOCK;
  float time1, time2, dub_time;
  float gpu_elapsed_time_ms;

  // Allocate memory space on the device
  REAL *d_a, *d_b, *d_c;
  hipMalloc((void **) &d_a, sizeof(REAL)*nn*nn);
  hipMalloc((void **) &d_b, sizeof(REAL)*nn*nn);
  hipMalloc((void **) &d_c, sizeof(REAL)*nn*nn);

  N_BLOCK=nn/32;

  dim3 dimGrid(N_BLOCK, N_BLOCK);
  dim3 dimBlock(nn/N_BLOCK, nn/N_BLOCK);

  printf("===============================\n");
  if(sizeof(a[1][1]) == sizeof(float))
    printf(" single precision\n");
  else
    printf("double precision\n");
  printf("size %d \n",nn );
  printf("#   block %d \n", N_BLOCK*N_BLOCK);
  printf("dim block %d \n", (nn/N_BLOCK)*(nn/N_BLOCK));
  printf("===============================\n");
  printf("Initialization\n");

  /* initialize matrix */
  time1 = clock();
  for (j = 0; j < nn; j++) {
    for (i = 0; i < nn; i++) {
      a[j][i] = ((REAL)rand())/((REAL)RAND_MAX);
      b[j][i] = ((REAL)rand())/((REAL)RAND_MAX);
      c[j][i] = 0.0L;		
      check[j][i] = 0.0L;		
    }
  }

  time2 = clock();
  dub_time = (time2 - time1)/(double) CLOCKS_PER_SEC;
  printf("Elapsed time for initialization \n");
  printf("Total time -----------------> %f \n", dub_time);

  gettimeofday(&start, 0);
  time1 = clock();

  // some events to count the execution time
  hipEvent_t custart, custop;
  hipEventCreate(&custart);
  hipEventCreate(&custop);
  hipEventRecord(custart, 0);
  hipEventSynchronize(custart);

  // copy matrix A and B from host to device memory
  hipMemcpy(d_a, a, sizeof(REAL)*nn*nn, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, sizeof(REAL)*nn*nn, hipMemcpyHostToDevice);

  gpu_mm <<< dimGrid, dimBlock >>> (d_a, d_b, d_c, nn);

  // Transfer results from device to host
  hipMemcpy(c, d_c, sizeof(REAL)*nn*nn, hipMemcpyDeviceToHost);

  // time counting terminate
  hipEventRecord(custop, 0);
  hipEventSynchronize(custop);

  time2 = clock();
  gettimeofday(&stop, 0);

  // compute time elapsed on GPU computing
  hipEventElapsedTime(&gpu_elapsed_time_ms, custart, custop);

  dub_time = (time2 - time1)/(double) CLOCKS_PER_SEC;

  // RESULTS
  sec    = stop.tv_sec  - start.tv_sec;
  usec   = stop.tv_usec - start.tv_usec;
  elapsed = sec + usec*1E-6;
  //
  printf("===============================\n");
  printf("Time   (1)-------------> %f \n", dub_time);
  printf("Mflops (1)-------------> %f \n", 
          2.0*nn*nn*nn/(1000*1000*dub_time));
  printf("Time   (2)-------------> %f \n", elapsed);
  printf("Mflops (2)-------------> %f \n", 
          2.0*nn*nn*nn/(1000*1000*elapsed));
  printf("Time   (3)-------------> %f \n", gpu_elapsed_time_ms/1000);
  printf("Mflops (3)-------------> %f \n", 
          2.0*nn*nn*nn/(1000*1000*gpu_elapsed_time_ms/1000));

  /* simple check */
  printf("Check -----------------> %f \n", c[nn/2][nn/2]);

#ifdef VALIDATION
  for (j = 0; j < nn; j++) { 
      for (k = 0; k < nn; k++) { 
          for (i = 0; i < nn; i++) { 
              check[j][i] = check[j][i]+a[j][k]*b[k][i];		
          } 
      } 
  } 
//
  for (j = 0; j < nn; j++) { 
      for (i = 0; i < nn; i++) { 
          printf("Error ---------> %lf, \n", check[j][i]-c[j][i]); 
      } 
  } 
#else
// do nothing
#endif

   return 0;  
}

